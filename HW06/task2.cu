#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

int main(int argc, char* argv[]) {
    // Deal with input values
    if (argc != 4) {
        std::cerr << "Wrong inputs";
        return -1;
    }
    unsigned int n = std::atoi(argv[1]);
    unsigned int R = std::atoi(argv[2]);
    unsigned int threads_per_block = std::atoi(argv[3]);

    // Initialize arrays for host
    float* h_image = new float[n];
    float* h_mask = new float[2 * R + 1];
    float* h_output = new float[n];

    // Fill matrices A and B with random numbers in the range [-1, 1]
    std::mt19937 generator(759);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (unsigned int i = 0; i < n; i++) {
        h_image[i] = dist(generator);
    }
    for (unsigned int i = 0; i < 2 * R + 1; i++) {
        h_mask[i] = dist(generator);
    }

    // Allocate device memory
    float *d_image, *d_mask, *d_output;
    hipMalloc(&d_image, n * sizeof(float));
    hipMalloc(&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_image, h_image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Call stencil function
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    // Copy result matrix back to host
    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the last element of matrix h_output
    std::cout << h_output[n - 1] << std::endl;

    // Print elapsed time
    std::cout << elapsed_time_ms << std::endl;

    // Free host memory
    delete[] h_image;
    delete[] h_mask;
    delete[] h_output;

    //Free device memory
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}